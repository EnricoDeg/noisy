#include "hip/hip_runtime.h"
/*
 * @file backendCUDAcomplex.cu
 *
 * @copyright Copyright (C) 2024 Enrico Degregori <enrico.degregori@gmail.com>
 *
 * @author Enrico Degregori <enrico.degregori@gmail.com>
 * 
 * MIT License
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions: 
 * 
 * The above copyright notice and this permission notice shall be included
 * in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
 * OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
 * HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
 * WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS IN THE SOFTWARE.
 */

#include "src/backend/cuda/backendCUDA.hpp"

#include "cuAlgo.hpp"

#include <cassert>

template<typename Tdata>
__global__ void corrComplexKernel(thrust::complex<Tdata> * __restrict__ dataIn1,
                                  thrust::complex<Tdata> * __restrict__ dataIn2,
                                  thrust::complex<Tdata> * __restrict__ dataOut,
                                  unsigned int size) {

	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < size) {

		dataOut[i] = dataIn1[i] * thrust::conj(dataIn2[i]);
		i += gridDim.x * blockDim.x;
	}
}

template<typename Tdata>
__global__ void convComplexKernel(thrust::complex<Tdata> * __restrict__ dataIn1,
                                  thrust::complex<Tdata> * __restrict__ dataIn2,
                                  thrust::complex<Tdata> * __restrict__ dataOut,
                                  unsigned int size) {

	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < size) {

		dataOut[i] = dataIn1[i] * dataIn2[i];
		i += gridDim.x * blockDim.x;
	}
}

template <typename Tdata>
void cuda_complex_impl<Tdata>::op::corrComplex(thrust::complex<Tdata> * __restrict__ dataIn1,
                                               thrust::complex<Tdata> * __restrict__ dataIn2,
                                               thrust::complex<Tdata> * __restrict__ dataOut,
                                               unsigned int size) {

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 blocksPerGrid(div_ceil(size, THREADS_PER_BLOCK));
    corrComplexKernel<Tdata><<<blocksPerGrid, threadsPerBlock>>>(dataIn1, dataIn2, dataOut, size);
    check_cuda( hipStreamSynchronize(0) );
}

template <typename Tdata>
void cuda_complex_impl<Tdata>::op::convComplex(thrust::complex<Tdata> * __restrict__ dataIn1,
                                               thrust::complex<Tdata> * __restrict__ dataIn2,
                                               thrust::complex<Tdata> * __restrict__ dataOut,
                                               unsigned int size) {

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 blocksPerGrid(div_ceil(size, THREADS_PER_BLOCK));
    convComplexKernel<Tdata><<<blocksPerGrid, threadsPerBlock>>>(dataIn1, dataIn2, dataOut, size);
    check_cuda( hipStreamSynchronize(0) );
}
