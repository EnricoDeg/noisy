/*
 * @file backendCUDAfourier.cu
 *
 * @copyright Copyright (C) 2024 Enrico Degregori <enrico.degregori@gmail.com>
 *
 * @author Enrico Degregori <enrico.degregori@gmail.com>
 * 
 * MIT License
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions: 
 * 
 * The above copyright notice and this permission notice shall be included
 * in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
 * OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
 * HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
 * WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS IN THE SOFTWARE.
 */

#include "src/backend/cuda/backendCUDAfourier.hpp"
#include "cuAlgo.hpp"

namespace cuda {

    namespace details {

        template<class T>
        class fft_execute { };

        template<>
        class fft_execute<float> {
            public:
            static void execute(hipfftHandle plan, hipfftComplex *idata, hipfftComplex *odata, int direction) {
                hipfftExecC2C(plan, (hipfftComplex *)idata, (hipfftComplex *)odata, direction);
            }
        };

        template<>
        class fft_execute<double> {
            public:
            static void execute(hipfftHandle plan, hipfftDoubleComplex *idata, hipfftDoubleComplex *odata, int direction) {
                hipfftExecZ2Z(plan, (hipfftDoubleComplex *)idata, (hipfftDoubleComplex *)odata, direction);
            }
        };

        template<typename T, typename ComplexT, hipfftType type>
        fourier_impl<T, ComplexT, type>::fourier_impl(unsigned int rows, unsigned int cols)
        : m_rows(rows),
          m_cols(cols)
        {

            hipfftPlan2d(&m_plan, rows, cols, type);
        }

        template<typename T, typename ComplexT, hipfftType type>
        fourier_impl<T, ComplexT, type>::~fourier_impl()
        {

            hipfftDestroy(m_plan);
        }

        template<typename T, typename ComplexT, hipfftType type>
        void fourier_impl<T, ComplexT, type>::fft(thrust::complex<T> * data) {

            fft_execute<T>::execute(m_plan,
                                    reinterpret_cast<ComplexT *>(data),
                                    reinterpret_cast<ComplexT *>(data),
                                    HIPFFT_FORWARD);
        }

        template<typename T, typename ComplexT, hipfftType type>
        void fourier_impl<T, ComplexT, type>::ifft(thrust::complex<T> * data) {

            fft_execute<T>::execute(m_plan,
                                    reinterpret_cast<ComplexT *>(data),
                                    reinterpret_cast<ComplexT *>(data),
                                    HIPFFT_BACKWARD);
        }

        template<typename T, typename ComplexT, hipfftType type>
        void fourier_impl<T, ComplexT, type>::fftshift(thrust::complex<T> * data) {

            cuAlgo::fftshift2dMatrix(data, m_rows, m_cols);
        }

        template class fourier_impl<float, hipfftComplex, HIPFFT_C2C>;

    }

}
