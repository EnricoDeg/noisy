#include "hip/hip_runtime.h"
/*
 * @file backendCUDAop.cu
 *
 * @copyright Copyright (C) 2024 Enrico Degregori <enrico.degregori@gmail.com>
 *
 * @author Enrico Degregori <enrico.degregori@gmail.com>
 * 
 * MIT License
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions: 
 * 
 * The above copyright notice and this permission notice shall be included
 * in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
 * OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
 * HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
 * WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS IN THE SOFTWARE.
 */

#include "src/backend/cuda/backendCUDA.hpp"

#include <cassert>

#include "cuAlgo.hpp"

template<typename T>
__global__ void sumInPlaceKernel(T            * __restrict__ data1,
                                 const T      * __restrict__ data2,
                                 unsigned int                size ) {

	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < size) {

		data1[i] += data2[i];
		i += gridDim.x * blockDim.x;
	}
}

template <typename Tdata>
void cuda_impl<Tdata>::op::normalize(Tdata * __restrict__ data, unsigned int size) {

    cuAlgo::normalizeVector(data, size);
}

template <typename Tdata>
void cuda_impl<Tdata>::op::fliplr(Tdata * __restrict__ data, unsigned int dim,
                                  unsigned int mRows, unsigned int mCols) {

    assert(dim == 0 || dim == 1);
    cuAlgo::fliplr1dMatrix(data, dim, mRows , mCols);
}

template <typename Tdata>
void cuda_impl<Tdata>::op::sumInPlace(Tdata * __restrict__ data1,
                                      const Tdata * __restrict__ data2,
                                      unsigned int size) {

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 blocksPerGrid(div_ceil(size, THREADS_PER_BLOCK));
    sumInPlaceKernel<Tdata><<<blocksPerGrid, threadsPerBlock>>>(data1, data2, size);
    check_cuda( hipStreamSynchronize(0) );
}
