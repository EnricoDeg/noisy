#include "hip/hip_runtime.h"
/*
 * @file backendCUDAop.cu
 *
 * @copyright Copyright (C) 2024 Enrico Degregori <enrico.degregori@gmail.com>
 *
 * @author Enrico Degregori <enrico.degregori@gmail.com>
 * 
 * MIT License
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions: 
 * 
 * The above copyright notice and this permission notice shall be included
 * in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
 * OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
 * HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
 * WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS IN THE SOFTWARE.
 */

#include "src/backend/cuda/backendCUDA.hpp"

#include <cassert>
#include <cmath>

#include "cuAlgo.hpp"

template<typename T>
__global__ void sumInPlaceKernel(T            * __restrict__ data1,
                                 const T      * __restrict__ data2,
                                 unsigned int                size ) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < size) {

        data1[i] += data2[i];
        i += gridDim.x * blockDim.x;
    }
}

template<typename T>
__global__ void prodInPlaceKernel(T            * __restrict__ data1,
                                  const T      * __restrict__ data2,
                                  unsigned int                size ) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < size) {

        data1[i] *= data2[i];
        i += gridDim.x * blockDim.x;
    }
}

template<typename T>
__global__ void divScalarInPlaceKernel(T            * __restrict__ data  ,
                                       unsigned int                size  ,
                                       T                           scalar) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < size) {

        data[i] /= scalar;
        i += gridDim.x * blockDim.x;
    }
}

template<typename T>
__global__ void mirrorKernel(T            * __restrict__ inData ,
                             T            * __restrict__ outData,
                             unsigned int                size   ) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < size) {

        outData[i] = inData[i] * std::pow(-1.0, i);
        i += gridDim.x * blockDim.x;
    }
}

template<typename T>
__global__ void applyThresholdKernel(T            * __restrict__ data     ,
                                     T                           threshold,
                                     unsigned int                size     ) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < size) {

        if (abs(data[i]) < abs(threshold))
            data[i] = 0;
        i += gridDim.x * blockDim.x;
    }
}

template <typename Tdata>
void cuda_impl<Tdata>::op::normalize(Tdata * __restrict__ data, unsigned int size) {

    cuAlgo::normalizeVector(data, size);
}

template <typename Tdata>
void cuda_impl<Tdata>::op::fliplr(Tdata * __restrict__ data, unsigned int dim,
                                  unsigned int mRows, unsigned int mCols) {

    assert(dim == 0 || dim == 1);
    cuAlgo::fliplr1dMatrix(data, dim, mRows , mCols);
}

template <typename Tdata>
void cuda_impl<Tdata>::op::sumInPlace(Tdata * __restrict__ data1,
                                      const Tdata * __restrict__ data2,
                                      unsigned int size) {

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 blocksPerGrid(div_ceil(size, THREADS_PER_BLOCK));
    sumInPlaceKernel<Tdata><<<blocksPerGrid, threadsPerBlock>>>(data1, data2, size);
    check_cuda( hipStreamSynchronize(0) );
}

template <typename Tdata>
void cuda_impl<Tdata>::op::prodInPlace(Tdata * __restrict__ data1,
                                       const Tdata * __restrict__ data2,
                                       unsigned int size) {

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 blocksPerGrid(div_ceil(size, THREADS_PER_BLOCK));
    prodInPlaceKernel<Tdata><<<blocksPerGrid, threadsPerBlock>>>(data1, data2, size);
    check_cuda( hipStreamSynchronize(0) );
}

template <typename Tdata>
void cuda_impl<Tdata>::op::divScalarInPlace(Tdata * __restrict__ data ,
                                            unsigned int         size ,
                                            Tdata                value) {

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 blocksPerGrid(div_ceil(size, THREADS_PER_BLOCK));
    divScalarInPlaceKernel<Tdata><<<blocksPerGrid, threadsPerBlock>>>(data, size, value);
    check_cuda( hipStreamSynchronize(0) );
}

template <typename Tdata>
void cuda_impl<Tdata>::op::mirror(Tdata * __restrict__ inData ,
                                  Tdata * __restrict__ outData,
                                  unsigned int         size   ) {

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 blocksPerGrid(div_ceil(size, THREADS_PER_BLOCK));
    mirrorKernel<Tdata><<<blocksPerGrid, threadsPerBlock>>>(inData, outData, size);
    check_cuda( hipStreamSynchronize(0) );
}

template <typename Tdata>
void cuda_impl<Tdata>::op::applyThreshold(Tdata        * __restrict__ inData   ,
                                          Tdata                       threshold,
                                          unsigned int                size     ) {

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 blocksPerGrid(div_ceil(size, THREADS_PER_BLOCK));
    applyThresholdKernel<Tdata><<<blocksPerGrid, threadsPerBlock>>>(inData, threshold, size);
    check_cuda( hipStreamSynchronize(0) );
}
