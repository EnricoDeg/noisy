#include "hip/hip_runtime.h"
/*
 * @file backendCUDAmemory.cu
 *
 * @copyright Copyright (C) 2024 Enrico Degregori <enrico.degregori@gmail.com>
 *
 * @author Enrico Degregori <enrico.degregori@gmail.com>
 * 
 * MIT License
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions: 
 * 
 * The above copyright notice and this permission notice shall be included
 * in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
 * OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
 * HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
 * WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS IN THE SOFTWARE.
 */

#include "src/backend/cuda/backendCUDA.hpp"

#include "cuAlgo.hpp"

template<typename T>
__global__ void fillKernel(T            * __restrict__ data,
                           unsigned int                size,
                           T                          value) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < size) {

        data[i] = value;
        i += gridDim.x * blockDim.x;
    }
}

template <typename Tdata>
Tdata * cuda_impl<Tdata>::memory::allocate(unsigned int elements) {
    Tdata *p;
    check_cuda( hipMalloc(&p, elements * sizeof(Tdata)) );
    return p;
}

template <typename Tdata>
void cuda_impl<Tdata>::memory::free(Tdata * data) {
    check_cuda( hipFree(data) );
}

template <typename Tdata>
void cuda_impl<Tdata>::memory::copy(Tdata *dst, Tdata *src, unsigned int size) {
    check_cuda( hipMemcpy(dst, src, size*sizeof(Tdata), hipMemcpyDeviceToDevice) );
}

template <typename Tdata>
void cuda_impl<Tdata>::memory::fill(Tdata * __restrict__ data, unsigned int size, Tdata value) {
    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 blocksPerGrid(div_ceil(size, THREADS_PER_BLOCK));
    fillKernel<Tdata><<<blocksPerGrid, threadsPerBlock>>>(data, size, value);
    check_cuda( hipStreamSynchronize(0) );
}
